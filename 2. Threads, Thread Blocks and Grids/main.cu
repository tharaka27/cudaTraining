#include <iostream>
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <ctime>
#include <chrono> 

using namespace std;

__global__ void AddMat(int* a, int* b, int count) {
 
	//int id = blockIdx.x * blockDim.x + threadIdx.x;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < count) {
		a[index] += b[index];
	}

}

int main() {

	int count = 30000000;
	int *h_a = new int[count];
	int *h_b = new int[count];
	int* h_c = new int[count];

	srand(time(NULL));
 

	for (int i = 0; i < count; i++) {
		h_a[i] = rand() % 1000;
		h_b[i] = rand() % 1000;
	
	}

	auto start = chrono::high_resolution_clock::now();
	for (int i = 0; i < count; i++) {
		h_c[i] = h_a[i] + h_b[i];
	}
	auto end = chrono::high_resolution_clock::now();
	double time_taken_CPU = chrono::duration_cast<chrono::nanoseconds>(end - start).count();




	for (int i = 0; i < 5; i++) {
		cout << h_a[i] << " + " << h_b[i] << endl;

	}

	int *d_a, *d_b;

	if (hipMalloc(&d_a, sizeof(int) * count) != hipSuccess) {
		cout << "Cound not allocate enough memory to d_a variable" << endl;
		return 0;
	}

	if (hipMalloc(&d_b, sizeof(int) * count) != hipSuccess) {
		cout << "Cound not allocate enough memory to d_a variable" << endl;
		hipFree(d_a);
		return 0;
	}


	if (hipMemcpy(d_a, h_a, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess) {
		cout << "Cound not copy memory to d_a variable" << endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}

	if (hipMemcpy(d_b, h_b, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess) {
		cout << "Cound not copy memory to d_b variable" << endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}
	
	start = chrono::high_resolution_clock::now();
	
	AddMat <<<count/256 + 1,256>>>(d_a, d_b, count);

	end = chrono::high_resolution_clock::now();
	double time_taken_GPU = chrono::duration_cast<chrono::nanoseconds>(end - start).count();


	if (hipMemcpy(h_a, d_a, sizeof(int) * count, hipMemcpyDeviceToHost) != hipSuccess) {
		cout << "Cound not copy memory to d_a variable" << endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}

	hipFree(d_a);
	hipFree(d_b);
	
	
	for (int i = 0; i < 5; i++) {
		cout << h_a[i] << endl;

	}

	cout << "Calculate value difference between CPU and GPU calculation" << endl;
	int notchange = 0;
	for (int i = 0; i < count; i++) {
		if (h_a[i] == h_c[i]) {
			notchange++;
		}
	}
	time_taken_CPU *= 1e-9;
	time_taken_GPU *= 1e-9;
	cout << "equal/total: " << notchange <<"/" << count << endl;
	cout << "CPU time: " << time_taken_CPU  <<"s" << endl;
	cout << "GPU time: " << time_taken_GPU  <<"s" << endl;
	cout << "Speed UP in GPU: " << time_taken_CPU/time_taken_GPU << "x times CPU"  << endl;

	free(h_a);
	free(h_b);
	return 0;
}